#include "hip/hip_runtime.h"
#include "pch.h"

__global__ void mmul( float *a, float *b, float *c, int m, int n, int q)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0;
    if( col < q && row < m)
    {
        for(int i = 0; i < n; i++)
        {
            sum += a[row*n+i] * b[i*q+col];
        }
        c[row*q+col] = sum;
    }
}



__host__ void gpuMult(float *h_A, float *h_B, float *gpu_C, const int m, const int n, const int p, const int q, const int block_size)
{
  //declare variables to be used by GPU (device) for matrix multiplication
  float *d_A, *d_B, *d_C;

  //Allocate device memory
  hipMalloc(&d_A, m*n*sizeof(float));
  hipMalloc(&d_B, p*q*sizeof(float));
  hipMalloc(&d_C, m*q*sizeof(float));
  cudaCheckErrors("hipMalloc failure");
  hipMemcpy(d_A, h_A, m*n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, p*q*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failture");
  
  //calculate grid and block dimensions
  unsigned int grid_rows = (m + block_size - 1) / block_size;
  unsigned int grid_cols = (q + block_size - 1) / block_size;
  dim3 grid(grid_cols, grid_rows);
  dim3 block(block_size, block_size);
 
  printf("Kernel launch dimensions: \n");
  printf("\tGrid size  : {%d, %d, %d} blocks.\n",grid.x, grid.y, grid.z);
  printf("\tBlock size : {%d, %d, %d} threads.\n",block.x, block.y, block.z);


  //carry out matrix multiplication on the GPUs
  mmul<<<grid,block>>>(d_A,d_B,d_C,m,n,q);
  hipDeviceSynchronize();
  cudaCheckErrors("Kernel execution failure or hipMemcpy H2D failure");
  

  // Transefr results from device to host 
  hipMemcpy(gpu_C, d_C, sizeof(float)*m*q, hipMemcpyDeviceToHost);

  // Cleanup
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  cudaCheckErrors("hipFree failure");
}
