#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "pch.h"

__global__ void SharedMatmul(const float *a, const float *b, float *c, const int m, const int p, const int q) {
  // Compute each thread's global row and column index.
  // int row = ???
  // int col = ???

  // Statically allocate a tile of shared memory. Tile size should equal the
  // number of threads per block.
  // ??? float s_a[???];
  // ??? float s_b[???];

  // Declare a temporary variable to accumulate calculated elements
  // for the C matrix.
  float tmp = 0.0;

  // Sweep tiles of size blockDim.x across matrices A and B.
  for (int i = 0; i < p; i += blockDim.x) {
   
    // Load in elements from A and B into shared memory into each tile.
    // int shared_index = ???

    // For matrix A, keep the row invariant and iterate through columns.
    // s_a[shared_index] = a[row * ??? + ??? + ???];

    // For matrix B, keep the column invariant and iterate through rows.
    // s_b[shared_index] = b[??? * ??? + ??? * ??? + col];

    // Wait for tiles to be loaded in before doing computation.

    // Do matrix multiplication on the small matrix within the current tile.
    // for (int j = 0; j < ???; j++) {
    //   tmp += s_a[??? * ??? + j] * s_b[j * ??? + ???];
    // }
 
    // Wait for all threads to finish using current tiles before loading in new ones.
  }

  // Write resulting calculations as elements of the C matrix.
  // c[row * q + col] = tmp;
}

__host__ void gpuMatmul(const float *h_A, const float *h_B, float *gpu_C, const int m, const int p, const int q)
{
  float *d_A, *d_B, *d_C;

  // Allocate device matrices on GPU using hipMalloc
  hipMalloc(&d_A, m*p*sizeof(float));
  hipMalloc(&d_B, p*q*sizeof(float));
  hipMalloc(&d_C, m*q*sizeof(float));
  cudaCheckErrors("hipMalloc failure");
  // Copy host matrices A and B to the device using hipMemcpy
  hipMemcpy(d_A, h_A, m*p*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, p*q*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failture");

  // Set threads per CUDA block dimension. The maximum number
  // of total threads is 1024.
  int thread_dim = BLOCK_SIZE;
  // Set blocks per grid dimension (assume thread_dim divides M and N evenly)
  int blocks_x = q / thread_dim;
  int blocks_y = m / thread_dim;
  // Use dim3 structs for block and grid dimensions
  dim3 threads(thread_dim, thread_dim);
  dim3 blocks(blocks_x, blocks_y);

  // Launch the kernel to calculate AxB=C on the device
  SharedMatmul<<<blocks, threads>>>(d_A, d_B, d_C, m, p, q);
  cudaCheckErrors("kernel launch failure");
  // Synchronize the device, then copy device's C matrix to the host
  hipDeviceSynchronize();
  hipMemcpy(gpu_C, d_C, m*q*sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckErrors("Kernel execution failure or hipMemcpy H2D failure");

  // Free the device matrices
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  cudaCheckErrors("hipFree failure");
}
