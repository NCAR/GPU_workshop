#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <ctime>
#include <hip/hip_runtime.h>
#include <mma.h>




#define M 16

#define M_TILES 1024

#define M_TOTAL (M * M_TILES)
#define WARP_SIZE 32
using namespace nvcuda;

__host__ void InitMatrix(half *A, half *B, half *C)
{
	for (int i = 0; i < M_TOTAL*M_TOTAL; i++)
		A[i] = __float2half(rand() % 1000 / 1000.0f);
	for (int i = 0; i < M_TOTAL*M_TOTAL; i++)
		B[i] = __float2half(rand() % 1000 / 1000.0f);
	for (int i = 0; i < M_TOTAL*M_TOTAL; i++)
		C[i] = __float2half(rand() % 1000 / 1000.0f);
}

__global__ void fma_tensor(half *A, half *B, half *C, half *D)
{
	int ix = (blockIdx.x * blockDim.x + threadIdx.x)/WARP_SIZE;
	int iy = (blockIdx.y * blockDim.y + threadIdx.y);

	wmma::fragment<wmma::matrix_a, M, M, M, half, wmma::row_major> a_frag;
	wmma::fragment<wmma::matrix_b, M, M, M, half, wmma::row_major> b_frag;
	wmma::fragment<wmma::accumulator, M, M, M, half> ab_frag;
	wmma::fragment<wmma::accumulator, M, M, M, half> c_frag;
	
	wmma::fill_fragment(ab_frag, __float2half(0.0f));

	// AB = A*B
	int a_col, a_row, b_col, b_row, c_col, c_row;
	a_row = ix * M;
	b_row = iy * M;
	for (int k=0; k<M_TOTAL; k+=M) {
		a_col = b_col = k;

		if (a_row < M_TOTAL && a_col < M_TOTAL && b_row < M_TOTAL && b_col < M_TOTAL) {
			// Load the inputs
			wmma::load_matrix_sync(a_frag, A + a_col + a_row * M_TOTAL, M_TOTAL);
			wmma::load_matrix_sync(b_frag, B + b_col + b_col * M_TOTAL, M_TOTAL);

			// Perform the matrix multiplication
			wmma::mma_sync(ab_frag, a_frag, b_frag, ab_frag);
		}
	}

	// D = AB + C
	c_col = b_row;
	c_row = a_row;
	if (c_row < M_TOTAL && c_col < M_TOTAL) {
		wmma::load_matrix_sync(c_frag, C + c_col + c_row * M_TOTAL, M_TOTAL, wmma::mem_row_major);

		for (int i = 0; i < c_frag.num_elements; i++) {
			c_frag.x[i] = ab_frag.x[i] + c_frag.x[i];
		}

		// Store the output
		wmma::store_matrix_sync(D + c_col + c_row * M_TOTAL, c_frag, M_TOTAL, wmma::mem_row_major);
	}
}


int main()
{
	half *h_A, *h_B, *d_A, *d_B;
	half *h_C, *h_D, *d_C, *d_D;

	h_A = new half[M_TOTAL*M_TOTAL];
	h_B = new half[M_TOTAL*M_TOTAL];
	h_C = new half[M_TOTAL*M_TOTAL];
	h_D = new half[M_TOTAL*M_TOTAL];
	
	InitMatrix(h_A, h_B, h_C);
	printf("Matrices are size:\n");
	printf("h_A: %d x %d\n", M_TOTAL, M_TOTAL);	
	printf("h_B: %d x %d\n", M_TOTAL, M_TOTAL);	
	printf("h_C: %d x %d\n", M_TOTAL, M_TOTAL);	
	int MSizeBytesHalf;
	
	MSizeBytesHalf = sizeof(half) * M_TOTAL * M_TOTAL;

	hipMalloc((void**)&d_A, MSizeBytesHalf);
	hipMalloc((void**)&d_B, MSizeBytesHalf);
	hipMalloc((void**)&d_C, MSizeBytesHalf);
	hipMalloc((void**)&d_D, MSizeBytesHalf);

	hipMemcpy(d_A, h_A, MSizeBytesHalf, hipMemcpyHostToDevice); 
	hipMemcpy(d_B, h_B, MSizeBytesHalf, hipMemcpyHostToDevice); 
	hipMemcpy(d_C, h_C, MSizeBytesHalf, hipMemcpyHostToDevice); 

	//Kernel Invoke Paramters (2D grid and blocks) 
	int dimx = 16; 
	int dimy = 16; 

	dim3 block(dimx, dimy); //Block of 256 threads 
	dim3 grid((M_TOTAL+block.x-1)/block.x, (M_TOTAL+block.y-1)/block.y); //grid dimensions 

	printf("Value of block %d \t %d \n",block.x,block.y);
	printf("Value of grid %d \t %d \n",grid.x,grid.y);

	// Record time using CUDA events
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);
	fma_tensor<<<block, grid>>>(d_A,d_B,d_C,d_D);
	hipEventRecord(stop);
	hipDeviceSynchronize(); 
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Tensor FMA execution took %f ms \n", milliseconds);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipMemcpy(h_D, d_D, MSizeBytesHalf, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipFree(d_D);

	delete[] h_A;
	delete[] h_B;
	delete[] h_C;
	delete[] h_D;

return 0;
}
